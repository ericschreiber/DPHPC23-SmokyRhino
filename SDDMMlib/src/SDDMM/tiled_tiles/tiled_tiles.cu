#include "hip/hip_runtime.h"
// here is what this GPU implementation does (and what it doesn't):
// - each thread block gets a row of the A matrix and its threads carry out all computations that involve this row. I do not do any load balancing here: if a row
//   has much more or less non-zero elements than another row this is not dealt with here.
// - the priority of this implementation is to KEEP AS MUCH AS POSSIBLE OF THE ROW THAT A BLOCK IS WORKING ON IN THE SHARED MEMORY OF THE BLOCK. if the full row
//   does not fit into shared mem then we resort to tiling.
//   - this explcit loading into the shared mem is necessary since otherwise the tiles of A would end up in L1 but data of B would also go there i.e. we could
//     not know if the data of A gets evicted from L1 but if we load it into shared mem (and nothing of B into shared mem) then we know that it will stay there.
// - the columns of B are just kept in GPU RAM and loaded from there when they are needed for the dot product i.e. we are not trying to keep them in some fast
//   memory. In case a thread has to work on more than one elem he will be assinged "consecutive" elements that correspond to a nonzero in C.
// - also, currently I am hardcoding this implementation to floats, so if we want to make it work with other datatypes we will need to change it in a few places.
//
// TILED TILES UPDATE: dont assing one col of B to a thread in a block but let all threads
// work on all blocks and then split the work between threads on the tile level. The working
// sets of the threads are not consecutive anymore but in regular intervals.

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <random>

#include "tiled_tiles/tiled_tiles.cuh"
#include "utils.h"

#define THREADS_PER_BLOCK 2
#define GPU_SHARED_MEM_SIZE_BYTES 8 + 32 * sizeof(float)                               // this is the size of shared mem on both the A100 and V100 GPUs.
                                                                                       // can force tiling (e.g. for testing) by setting this to something small.
#define COMPUTATION_SHARED_MEM_BYTES (GPU_SHARED_MEM_SIZE_BYTES - 32 * sizeof(float))  // reserve 32 floats for last reduction in tiled_dot_product_thread_subset
#define COMPUTATION_SHARED_MEM (COMPUTATION_SHARED_MEM_BYTES / sizeof(float))          // unit: floats

// std::count can't be used on the GPU
__device__ int count(const int* arr, int len, const int row_index)
{
    int count = 0;
    for (int i = 0; i < len; i++)
    {
        if (arr[i] == row_index)
        {
            count++;
        }
    }
    return count;
}

// helper function that abstracts away the indexing logic of computing a tiled dot product
// in the updated version, a thread does not compute the whole dot product of the tile but only a subset of it
__device__ float tiled_dot_product_thread_subset(
    const float* tile,
    const int tiling_step,
    const int normal_tile_size,
    const int curr_tile_size,  // curr_tile_size can be smaller than normal_tile_size if we are working on the last tile of a row
    const float* matrixB_transposed_GPU_values,
    const int B_col_index,  // indexes the col of B that we are computing the dot prod with in this method call
    const int k)            // number of rows of B
{
    const float* B_col_beginning = matrixB_transposed_GPU_values + B_col_index * k;        // ptr to start of col of B that we are computing the dot prod with
    const float* B_col_tile_beginning = B_col_beginning + tiling_step * normal_tile_size;  // don't want entire column but only a tile of it
    float sum_of_chunks = 0;
    int numChunksInTile = ceilf((float)curr_tile_size / 4);  // ceil needed in case the current tile is smaller than 4
    for (int i = threadIdx.x; i < numChunksInTile - 1; i += blockDim.x)
    {
        // compute the chunk of the dot product that this thread is responsible for
        const float* vector1_beginning = tile + i * 4;
        const float* vector2_beginning = B_col_tile_beginning + i * 4;
        sum_of_chunks += vector1_beginning[0] * vector2_beginning[0];
        sum_of_chunks += vector1_beginning[1] * vector2_beginning[1];
        sum_of_chunks += vector1_beginning[2] * vector2_beginning[2];
        sum_of_chunks += vector1_beginning[3] * vector2_beginning[3];
    }
    // let thread 0 take care of the last chunk (bc it might be smaller than 4)
    if (threadIdx.x == 0)
    {
        int last_chunk_size = curr_tile_size % 4;
        if (last_chunk_size != 0)
        {
            const float* vector1_beginning = tile + (numChunksInTile - 1) * 4;
            const float* vector2_beginning = B_col_tile_beginning + (numChunksInTile - 1) * 4;
            for (int i = 0; i < last_chunk_size; i++)
            {
                // cant unroll here because we only know last_chunk_size at runtime
                sum_of_chunks += vector1_beginning[i] * vector2_beginning[i];
            }
        }
    }

    __syncthreads();  // all threads wait togehter here before we reduce their results

    // warp-wide reduction
    unsigned mask = __activemask();
    for (int offset = warpSize / 2; offset > 0; offset /= 2)
    {
        sum_of_chunks += __shfl_down_sync(mask, sum_of_chunks, offset);
    }

    // reduce the result of the warp-wide reduction to a single value
    extern __shared__ float reduction_space[32 * sizeof(float)];  // in cuda we allocate shared mem in bytes
    if (threadIdx.x % warpSize == 0)
    {
        reduction_space[threadIdx.x / warpSize] = sum_of_chunks;
    }

    __syncthreads();

    // thread 0 sweeps over the values and returns the sum.
    // I guess this could also be done in a tree like fashion but log_2(32) is 5 (and 5 vs 32 steps shouldnt make a big difference I think).
    if (threadIdx.x == 0)
    {
        float fresh_sum = 0;
        for (int i = 0; i < warpSize; i++)
        {
            fresh_sum += reduction_space[i];
        }
        return fresh_sum;
    }
}

// helper function that abstracts away the indexing logic of grabbing the factor from C/writing result into result matrix (both are sparse matrices)
__device__ void elem_compute(
    float* tile,                 // ptr to tile in shared mem
    const int tiling_step,       // index of the current tile (in the set of all tiles)
    const int normal_tile_size,  // normal_tile_size is the size of a tile that is not the last tile of a row
    const int curr_tile_size,    // curr_tile_size can be smaller than normal_tile_size if we are working on the last tile of a row
    const float* matrixB_transposed_GPU_values,
    const float* matrixC_GPU_values,
    float* matrixResult_GPU_values,
    const int* matrixC_GPU_col_indices,
    const int k,       // number of rows of B
    const int offset)  // offset into matrixC_GPU_values and matrixResult_GPU_values
{
    float dot_prod = tiled_dot_product_thread_subset(
        tile,
        tiling_step,
        normal_tile_size,
        curr_tile_size,
        matrixB_transposed_GPU_values,
        matrixC_GPU_col_indices[offset],  // col of B for dot product is the same col in which the nonzero of C sits
        k);
    if (threadIdx.x == 0)
    {
        // no need for atomic add since only thread 0 is writing back (all the partial sums from the other thread have already been reduced)
        matrixResult_GPU_values[offset] += dot_prod * matrixC_GPU_values[offset];
    }
}

// this is the kernel function.
// assumes matrixB_transposed_GPU_values is transposed.
__global__ void tiled_tiles(
    const int k,  // number of rows of B
    const int numElementsC,
    const float* __restrict__ const matrixA_GPU_values,
    const float* __restrict__ const matrixB_transposed_GPU_values,
    const float* __restrict__ const matrixC_GPU_values,
    const int* __restrict__ const matrixC_GPU_row_indices,
    const int* __restrict__ const matrixC_GPU_col_indices,
    float* __restrict__ const matrixResult_GPU_values,
    const int* prevBlocksWorkAll,
    const int* tiles_sizes,
    const int tiling_steps)
{
    ////////////////    SETUP NECESSARY VARS    ////////////////
    int row_index = blockIdx.x;                                            // holds bc we have set up one block per row so n-th block will take on n-th row of A
    const float* A_vals_row_start = matrixA_GPU_values + (row_index * k);  // pointer to beginning of row of A that this thread block is working on.
    int prevBlocksWork = prevBlocksWorkAll[blockIdx.x];
    int nnzs = prevBlocksWorkAll[row_index + 1] - prevBlocksWorkAll[row_index];  // number of nnzs in this row of C (= amount of work for thread block)

    ////////////////    MAIN LOOP    ////////////////
    for (int tiling_step = 0; tiling_step < tiling_steps; tiling_step++)
    {
        ////////////////    COMPUTE SIZE OF CURR TILE    ////////////////
        int curr_tile_size = tiles_sizes[tiling_step];

        ////////////////    THREAD 0: COPY TILE INTO SHARED MEM    ////////////////
        // TODO: this can very likely also be parallelized over the threads in the block
        // decalare a ptr to a shared mem region (this needs to be done so that threads other than thread 0 can access the tile later on)
        extern __shared__ float tile[COMPUTATION_SHARED_MEM_BYTES];
        if (threadIdx.x == 0)
        {
            // copy the tile into shared mem (I think this copying happens float by float (bc of pointer arithmetic) but maybe also byte by byte (?))
            for (int i = 0; i < curr_tile_size; i++)
            {
                // second summand (in parentheses) = offset of the tile that we're working on in this iteration of outer loop
                tile[i] = *(A_vals_row_start + (tiling_step * COMPUTATION_SHARED_MEM) + i);
            }
        }

        __syncthreads();  // this is a barrier

        ////////////////    ACTUAL COMPUTATION    ////////////////
        for (int elem_index = 0; elem_index < nnzs; elem_index++)  // iterate over all elems OF THE ENTIRE ROW OF C (that this block is working on)
        {
            int offset = prevBlocksWork + elem_index;
            elem_compute(
                tile,
                tiling_step,
                COMPUTATION_SHARED_MEM,
                curr_tile_size,
                matrixB_transposed_GPU_values,
                matrixC_GPU_values,
                matrixResult_GPU_values,
                matrixC_GPU_col_indices,
                k,
                offset);
        }
    }
}

// precompute stuff that the main kernels need with 1 thread (doing it inside of the main kernel would make them all do the same precomputations)
__global__ void precomputation(
    const int numElementsC,
    const int* __restrict__ const matrixC_GPU_row_indices,
    int* prevBlocksWork,
    int numBlocks,
    int* tiles_sizes,
    int tiling_steps,
    int row_mem_size)
{
    // populate prevBlocksWork
    for (int i = 0; i < numBlocks; i++)
    {
        int last = 0;
        if (i != 0)
            last = prevBlocksWork[i];
        int counter = count(matrixC_GPU_row_indices, numElementsC, i);
        prevBlocksWork[i + 1] += last + counter;
    }

    // populate tiles_sizes
    for (int i = 0; i < tiling_steps; i++)
    {
        tiles_sizes[i] = COMPUTATION_SHARED_MEM;
        // last tile might be smaller than the regular tile size
        if (i == tiling_steps - 1 && row_mem_size % COMPUTATION_SHARED_MEM != 0)
        {
            tiles_sizes[i] = row_mem_size % COMPUTATION_SHARED_MEM;
        }
    }
}

// this is the function that is called from the outside and that launches the calls to the kernel function
void compute(
    const int m,
    const int n,
    const int k,  // number of rows of B
    const int numElementsC,
    const float* __restrict__ const matrixA_GPU_values,
    const float* __restrict__ const matrixB_transposed_GPU_values,
    const float* __restrict__ const matrixC_GPU_values,
    const int* __restrict__ const matrixC_GPU_row_indices,
    const int* __restrict__ const matrixC_GPU_col_indices,
    float* __restrict__ const matrixResult_GPU_values)
{
    int blocks = m;  // one block per row of A
    // allocate array that will be populated by the precomputation kernel
    int* prevBlocksWork;
    int row_mem_size = k * sizeof(float);                                           // size of a row of A (= non-sparse) in mem
    int tiling_steps = ceil(row_mem_size / (float)(COMPUTATION_SHARED_MEM_BYTES));  // #pieces that we need to chop row of A into (bc it might not fit into shared mem)
    int* tiles_sizes;
    CUDA_CHECK(hipMalloc((void**)&prevBlocksWork, (blocks + 1) * sizeof(int)));  // + 1 needed for the computation (for last block) of nnzs in the main kernel
    CUDA_CHECK(hipMalloc((void**)&tiles_sizes, tiling_steps * sizeof(int)));
    // run the precomputation kernel
    precomputation<<<1, 1>>>(numElementsC, matrixC_GPU_row_indices, prevBlocksWork, blocks, tiles_sizes, tiling_steps, row_mem_size);

    dim3 threadsPerBlock(THREADS_PER_BLOCK);

    // call main kernel
    // TODO: currently I am spawning dynamic shared mem, maybe non dynamic shared mem is better?
    tiled_tiles<<<blocks, threadsPerBlock, GPU_SHARED_MEM_SIZE_BYTES>>>(
        k,
        numElementsC,
        matrixA_GPU_values,
        matrixB_transposed_GPU_values,
        matrixC_GPU_values,
        matrixC_GPU_row_indices,
        matrixC_GPU_col_indices,
        matrixResult_GPU_values,
        prevBlocksWork,
        tiles_sizes,
        tiling_steps);
    // Aggregate the return value of the kernel
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // free the array prevBlocksWork on GPU
    CUDA_CHECK(hipFree(prevBlocksWork));
    CUDA_CHECK(hipFree(tiles_sizes));
}