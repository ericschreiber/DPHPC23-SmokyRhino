// better_naive_CSR_SDDMM_GPU.cpp

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <better_naive_CSR_GPU/better_naive_CSR_SDDMM_GPU.cuh>
#include <better_naive_CSR_GPU/better_naive_CSR_SDDMM_GPU_kernel.cuh>
#include <iostream>
#include <type_traits>
#include <typeinfo>

#include "utils.h"

void better_naive_CSR_SDDMM_GPU<float>::SDDMM_CSR(
    const DenseMatrix<float>& matrixA_HOST,
    const DenseMatrix<float>& matrixB_HOST,
    const CSRMatrix<float>& matrixC_HOST,
    CSRMatrix<float>& matrixResult_sparse_HOST,
    const int num_iterations) const
{
    // start the profiler
    // CUDA_CHECK(hipProfilerStart());
    // transpose matrixB to B^t
    DenseMatrix<float> matrixB_transpose_HOST = DenseMatrix<float>(matrixB_HOST);
    matrixB_transpose_HOST.transpose();

    // get sizes of matrixA and matrixB {A=mxk; B=kxn; B_transpose=nxk}
    int m = matrixA_HOST.getNumRows();
    int k = matrixA_HOST.getNumCols();
    int n = matrixB_transpose_HOST.getNumRows();
    int nnz = matrixC_HOST.getNumValues();

    int k_aligned = k;
    if (k % 4 != 0)
    {
        k_aligned = k + (4 - (k % 4));
    }
    assert(k_aligned % 4 == 0 && "Error: k_aligned is not a multiple of 4");
    int k_aligned_by_4 = k_aligned >> 2;

    // check the dimensions of the matrices
    assert(matrixB_transpose_HOST.getNumCols() == k && "Error: matrixB_transpose has incompatible dimensions");
    assert(matrixC_HOST.getNumRows() == m && "Error: matrixC has incompatible dimensions m");
    assert(matrixC_HOST.getNumCols() == n && "Error: matrixC has incompatible dimensions n");
    assert(matrixResult_sparse_HOST.getNumRows() == m && "Error: matrixResult has incompatible dimensions m");
    assert(matrixResult_sparse_HOST.getNumCols() == n && "Error: matrixResult has incompatible dimensions n");

    int L2_size = 6291456;
    int lines_per_block = L2_size / (16 * 80 * k);

    assert(lines_per_block > 0 && "Error: k is too big, we cannot calculate with such big k.");

    int warps_per_line = 32 / lines_per_block;
    assert(warps_per_line <= 32 && "Error: That's more warps per line, than we can do.");

    // allocate memory for the matrices on the GPU
    float* matrixA_GPU;
    float* matrixB_transpose_GPU;
    float* matrixResult_GPU;
    int* col_idx_GPU;
    int* row_ptr_GPU;
    CUDA_CHECK(
        hipMalloc(
            &matrixA_GPU,
            m * k_aligned * sizeof(float)));
    CUDA_CHECK(
        hipMalloc(
            &matrixB_transpose_GPU,
            n * k_aligned * sizeof(float)));
    CUDA_CHECK(
        hipMalloc(
            &matrixResult_GPU,
            nnz * sizeof(float)));
    CUDA_CHECK(
        hipMalloc(
            &col_idx_GPU,
            nnz * sizeof(int)));
    CUDA_CHECK(
        hipMalloc(
            &row_ptr_GPU,
            (m + 1) * sizeof(int)));

    // copy matrices to the GPU
    for (int i = 0; i < m; i++)
    {
        float temp[k_aligned];
        for (int j = 0; j < k; j++)
        {
            temp[j] = matrixA_HOST.getValues()[i * k + j];
        }
        for (int j = k; j < k_aligned; j++)
        {
            temp[j] = 0;
        }
        CUDA_CHECK(
            hipMemcpy(
                matrixA_GPU + i * k_aligned,
                temp,
                k_aligned * sizeof(float),
                hipMemcpyHostToDevice));
    }
    // CUDA_CHECK(
    //     hipMemcpy(
    //         matrixA_GPU,
    //         matrixA_HOST.getValues(),
    //         m * k * sizeof(float),
    //         hipMemcpyHostToDevice));
    for (int i = 0; i < n; i++)
    {
        float temp[k_aligned];
        for (int j = 0; j < k; j++)
        {
            temp[j] = matrixB_transpose_HOST.getValues()[i * k + j];
        }
        for (int j = k; j < k_aligned; j++)
        {
            temp[j] = 0;
        }
        CUDA_CHECK(
            hipMemcpy(
                matrixB_transpose_GPU + i * k_aligned,
                temp,
                k_aligned * sizeof(float),
                hipMemcpyHostToDevice));
    }
    // CUDA_CHECK(
    //     hipMemcpy(
    //         matrixB_transpose_GPU,
    //         matrixB_transpose_HOST.getValues(),
    //         n * k * sizeof(float),
    //         hipMemcpyHostToDevice));
    CUDA_CHECK(
        hipMemcpy(
            col_idx_GPU,
            (matrixC_HOST.getColIndices()).data(),
            nnz * sizeof(int),
            hipMemcpyHostToDevice));
    CUDA_CHECK(
        hipMemcpy(
            row_ptr_GPU,
            (matrixC_HOST.getRowArray()).data(),
            (m + 1) * sizeof(int),
            hipMemcpyHostToDevice));

    for (int i = 0; i < num_iterations; i++)
    {
        // start the timer
        this->start_run();

        // Call the kernel to execute the acutal SDDMM
        compute_blockwise(
            lines_per_block,
            warps_per_line,
            m,
            k_aligned_by_4,
            matrixA_GPU,
            matrixB_transpose_GPU,
            row_ptr_GPU,
            col_idx_GPU,
            matrixResult_GPU);

        // stop the timer
        this->stop_run();
    }

    // std::cout << "Run complete" << std::endl;
    // copy result from the GPU to the CPU
    float* return_values = new float[nnz];
    // std::cout << "nnz = " << nnz << std::endl;

    CUDA_CHECK(
        hipMemcpy(
            return_values,
            matrixResult_GPU,
            nnz * sizeof(float),
            hipMemcpyDeviceToHost));

    // Convert pointer to std::vector
    std::vector<float> result_vector(return_values, return_values + nnz);

    // set the result matrix
    matrixResult_sparse_HOST.setValues(result_vector);
    matrixResult_sparse_HOST.setColIndices(matrixC_HOST.getColIndices());
    matrixResult_sparse_HOST.setRowArray(matrixC_HOST.getRowArray());

    // free memory on the device and destroy the handle
    CUDA_CHECK(
        hipFree(
            matrixA_GPU));
    CUDA_CHECK(
        hipFree(
            matrixB_transpose_GPU));
    CUDA_CHECK(
        hipFree(
            matrixResult_GPU));
    CUDA_CHECK(
        hipFree(
            col_idx_GPU));
    CUDA_CHECK(
        hipFree(
            row_ptr_GPU));

    // stop the profiler
    // CUDA_CHECK(hipProfilerStop());

    return;
}

void better_naive_CSR_SDDMM_GPU<float>::SDDMM(
    const DenseMatrix<float>& matrixA_HOST,
    const DenseMatrix<float>& matrixB_HOST,
    const SparseMatrix<float>& matrixC_HOST,
    SparseMatrix<float>& matrixResult_HOST,
    const int num_iterations) const
{
    const CSRMatrix<float>* csrMatrixC = dynamic_cast<const CSRMatrix<float>*>(&matrixC_HOST);
    CSRMatrix<float>* csrMatrixResult = dynamic_cast<CSRMatrix<float>*>(&matrixResult_HOST);
    if (csrMatrixC == nullptr || csrMatrixResult == nullptr)
    {
        throw std::invalid_argument("Error: convert Sparse to CSR before using this function");
    }
    else
    {
        SDDMM_CSR(
            matrixA_HOST,
            matrixB_HOST,
            *csrMatrixC,
            *csrMatrixResult,
            num_iterations);
    }
}

template <typename T>
void better_naive_CSR_SDDMM_GPU<T>::SDDMM(
    const DenseMatrix<T>& matrixA_HOST,
    const DenseMatrix<T>& matrixB_HOST,
    const SparseMatrix<T>& matrixC_HOST,
    SparseMatrix<T>& matrixResult_HOST,
    const int num_iterations) const
{
    assert(false && "Error: better_naive_CSR_SDDMM_GPU::SDDMM() only accepts float as input. Other types are not supported");
}

void better_naive_CSR_SDDMM_GPU<float>::start_run() const
{
    assert(this->_timer != nullptr && "Error: better_naive_CSR_SDDMM_GPU::start_run() timer is nullptr. Check that you have set the timer with <SDDMM>.set_timer()");
    this->_timer->start_gpu_run();
}

void better_naive_CSR_SDDMM_GPU<float>::stop_run() const
{
    this->_timer->stop_gpu_run();
}

// Explicit template instantiation
// template class better_naive_CSR_SDDMM_GPU<float>;
template class better_naive_CSR_SDDMM_GPU<double>;
template class better_naive_CSR_SDDMM_GPU<int>;
