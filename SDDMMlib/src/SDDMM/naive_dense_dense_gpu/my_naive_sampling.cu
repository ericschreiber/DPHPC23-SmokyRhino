#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <algorithm>
#include <iostream>

#include "utils.h"

////
// this kernels computes the Hadamard Product of two dense matrices A and B
// A (m x n)
// B (m x n)
//
// the kernel expects the number of elements of the matrices as int (m*n) and
// pointers to the 2 matrices of type float in the following order A, B
// all pointers need to point to memory on the GPU
//
// the result is written to B
////

__global__ void naivesampling(
    const int size,
    const float *const A,
    float *const B)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        B[idx] = B[idx] * A[idx];
    }
}

void my_naive_sampling(
    const int size,
    const float *const A,
    float *const B)
{
    // every block can have up to 1024 threads
    int blocks = std::min(1024, (size + 1023) / 1024);
    std::cout << "blocks: " << blocks << std::endl;
    std::cout << "size: " << size << std::endl;
    std::cout << "Pointer A: " << A << std::endl;
    std::cout << "Pointer B: " << B << std::endl;
    std::cout << "Pointer to size: " << &size << std::endl;

    // Copy A to Host
    float *A_host = new float[size];
    CUDA_CHECK(hipMemcpy(A_host, A, size * sizeof(float), hipMemcpyDeviceToHost));
    // Print A
    std::cout << "A: ";
    for (int i = 0; i < size; i++)
    {
        std::cout << A_host[i] << " ";
    }
    std::cout << std::endl;

    // Copy B to Host
    float *B_host = new float[size];
    CUDA_CHECK(hipMemcpy(B_host, B, size * sizeof(float), hipMemcpyDeviceToHost));
    // Print B
    std::cout << "B: ";
    for (int i = 0; i < size; i++)
    {
        std::cout << B_host[i] << " ";
    }
    std::cout << std::endl;

    // naivesampling<<<blocks, 1024>>>(
    //     size,
    //     A,
    //     B);
    CUDA_CHECK(hipGetLastError());

    // synchronization not needed if implicit in the time measurement
    CUDA_CHECK(hipDeviceSynchronize());
}